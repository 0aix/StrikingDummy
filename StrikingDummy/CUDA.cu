#include "hip/hip_runtime.h"
#include "CUDA.cuh"
#include <iostream>
#include <chrono>
#include <math.h>
#include <hiprand.h>
#include <hipblas.h>

int blockSize = 0;
bool cuda_init = false;
hiprandGenerator_t gen;
hipblasHandle_t handle;

void cudaSafeDeviceSynchronize()
{
	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipDeviceSynchronize failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void cudaInitialize()
{
	if (!cuda_init)
	{
		hipError_t cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess)
		{
			std::cerr << "hipSetDevice failed: " << hipGetErrorString(cudaStatus) << std::endl;
			throw 0;
		}

		hipDeviceProp_t prop;
		cudaStatus = hipGetDeviceProperties(&prop, 0);
		if (cudaStatus != hipSuccess)
		{
			std::cerr << "hipGetDeviceProperties failed: " << hipGetErrorString(cudaStatus) << std::endl;
			throw 0;
		}
		blockSize = prop.maxThreadsPerBlock;
		if (blockSize != 1024)
			throw 0;

		hiprandStatus_t status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);
		if (status != HIPRAND_STATUS_SUCCESS)
		{
			std::cerr << "hiprandCreateGenerator failed" << std::endl;
			throw 0;
		}

		status = hiprandSetPseudoRandomGeneratorSeed(gen, std::chrono::high_resolution_clock::now().time_since_epoch().count());
		if (status != HIPRAND_STATUS_SUCCESS)
		{
			std::cerr << "hiprandSetPseudoRandomGeneratorSeed failed" << std::endl;
			throw 0;
		}

		hipblasStatus_t hipblasStatus_t = hipblasCreate(&handle);
		if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		{
			std::cerr << "hipblasCreate failed" << std::endl;
			throw 0;
		}

		cuda_init = true;
	}
}

void cudaSafeMalloc(void** A, int n)
{
	hipError_t cudaStatus = hipMalloc(A, n);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void cudaSafeFree(void** A)
{
	if (A)
	{
		hipFree(*A);
		*A = NULL;
	}
}

void cudaCopyToDevice(void* _A, void* A, int n)
{
	hipError_t cudaStatus = hipMemcpy(_A, A, n, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMemcpy failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

__global__ void _matrixInitialize(float* A, int n, float r)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		A[i] = 2.0f * (A[i] - 0.5f) * r;
}

void matrixInitialize(float** A, int n, int m)
{
	hipError_t cudaStatus = hipMalloc(A, sizeof(float) * n * m);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void matrixInitialize(float** A, int n, int m, float r)
{
	matrixInitialize(A, n, m);

	hiprandStatus_t status = hiprandGenerateUniform(gen, *A, (size_t)n * (size_t)m);
	if (status != HIPRAND_STATUS_SUCCESS)
	{
		std::cerr << "hiprandGenerateUniform failed" << std::endl;
		throw 0;
	}

	int numBlocks = (n * m + blockSize - 1) / blockSize;
	_matrixInitialize<<<numBlocks, blockSize>>>(*A, n * m, r);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_matrixInitialize failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void matrixInitialize(float* A, int n, int m, float r)
{
	hiprandStatus_t status = hiprandGenerateUniform(gen, A, (size_t)n * (size_t)m);
	if (status != HIPRAND_STATUS_SUCCESS)
	{
		std::cerr << "hiprandGenerateUniform failed" << std::endl;
		throw 0;
	}

	int numBlocks = (n * m + blockSize - 1) / blockSize;
	_matrixInitialize<<<numBlocks, blockSize>>>(A, n * m, r);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_matrixInitialize failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void matrixFree(float** A)
{
	if (A)
	{
		hipFree(*A);
		*A = NULL;
	}
}

void matrixMultiply(float* C, float* A, int n0, int m0, float* B, int n1, int m1)
{
	if (m0 != n1)
		throw 0;

	float a = 1.0f;
	float b = 0.0f;

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n0, m1, m0, &a, A, n0, B, n1, &b, C, n0);
}

void matrixMultiplyTranspose(float* C, float* A, int n0, int m0, float* B, int n1, int m1)
{
	if (m0 != n1)
		throw 0;

	float a = 1.0f;
	float b = 0.0f;

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n0, m1, m0, &a, A, n0, B, m1, &b, C, n0);
}

void matrixTransposeMultiply(float* C, float* A, int n0, int m0, float* B, int n1, int m1)
{
	if (m0 != n1)
		throw 0;

	float a = 1.0f;
	float b = 0.0f;

	hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n0, m1, m0, &a, A, m0, B, n1, &b, C, n0);
}

__global__ void _arrayInitialize(float* A, int n, float r)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		A[i] = r;
}

void arrayInitialize(float** A, int n, float r)
{
	hipError_t cudaStatus = hipMalloc(A, sizeof(float) * n);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}

	int numBlocks = (n + blockSize - 1) / blockSize;
	_arrayInitialize<<<numBlocks, blockSize>>>(*A, n, r);
}

__global__ void _arrayAddRepSigmoid(float* C, float* A, float* B, int n, int width)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		C[i] = 1.0f / (1.0f + expf(-(A[i] + B[i % width])));
}

__global__ void _arrayMultiplyDerivSigmoid(float* C, float* A, float* B, int n)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		C[i] = A[i] * B[i] * (1.0f - B[i]);
}

void arrayCopyToDevice(float* _A, float* A, int n)
{
	hipError_t cudaStatus = hipMemcpy(_A, A, sizeof(float) * n, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMemcpy failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void arrayCopyToHost(float* A, float* _A, int n)
{
	//hipError_t cudaStatus = hipMemcpy(A, _A, sizeof(float) * n, hipMemcpyDeviceToHost);
	hipError_t cudaStatus = hipMemcpyAsync(A, _A, sizeof(float) * n, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMemcpy failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void arrayAddRepSigmoid(float* C, float* A, float* B, int n, int m)
{
	int numBlocks = (n * m + blockSize - 1) / blockSize;
	_arrayAddRepSigmoid<<<numBlocks, blockSize>>>(C, A, B, n * m, n);
}

void arrayMultiplyDerivSigmoid(float* C, float* A, float* B, int n)
{
	int numBlocks = (n + blockSize - 1) / blockSize;
	_arrayMultiplyDerivSigmoid<<<numBlocks, blockSize>>>(C, A, B, n);
}

void arrayStep(float* B, float* A, float nu, int n)
{
	hipblasSaxpy(handle, n, &nu, A, 1, B, 1);
}

__global__ void _unpotato(float* A, int* B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ((i + 1) % 2500)
		A[i * 20 + B[i]] = 0.0f;
}

__global__ void _potato(float* A, float* B, unsigned char* C, float* D, int* E)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ((i + 1) % 2500)
	{
		// _d3, _X3, _action + 20 * offset, _reward + 2 * offset
		unsigned int m = i * 20;
		unsigned int n = m + 20;
		float q = 0.0f;
		/*
		for (int k = 0; k < 20; ++k)
			if (C[m + k] && B[n + k] > q)
				q = B[n + k];
		*/
		int k = 0;
		unsigned char a;
		while ((a = C[m + k++]) < 20)
			if (B[n + a] > q)
				q = B[n + a];
		float x = B[m + E[i]];
		A[m + E[i]] = (x - (D[i * 2] + D[i * 2 + 1] * q)) * x * (1.0f - x);
	}
}

void unpotato(float* A, int* B, int n)
{
	if (n % 1000)
		throw 0;
	_unpotato<<<n / 1000, 1000>>>(A, B);
}

void potato(float* A, float* B, unsigned char* C, float* D, int* E, int n)
{
	if (n % 1000)
		throw 0;
	_potato<<<n / 1000, 1000>>>(A, B, C, D, E);
}
