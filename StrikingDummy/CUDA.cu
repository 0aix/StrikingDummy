#include "hip/hip_runtime.h"
#include "CUDA.cuh"
#include <iostream>
#include <chrono>
#include <math.h>
#include <hiprand.h>
#include <hipblas.h>

#define NUM_ACTIONS 21

int blockSize = 0;
bool cuda_init = false;
hiprandGenerator_t gen;
hipblasHandle_t handle;

void cudaSafeDeviceSynchronize()
{
	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipDeviceSynchronize failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void cudaInitialize()
{
	if (!cuda_init)
	{
		hipError_t cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess)
		{
			std::cerr << "hipSetDevice failed: " << hipGetErrorString(cudaStatus) << std::endl;
			throw 0;
		}

		hipDeviceProp_t prop;
		cudaStatus = hipGetDeviceProperties(&prop, 0);
		if (cudaStatus != hipSuccess)
		{
			std::cerr << "hipGetDeviceProperties failed: " << hipGetErrorString(cudaStatus) << std::endl;
			throw 0;
		}
		blockSize = prop.maxThreadsPerBlock;
		if (blockSize != 1024)
			throw 0;

		hiprandStatus_t status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);
		if (status != HIPRAND_STATUS_SUCCESS)
		{
			std::cerr << "hiprandCreateGenerator failed" << std::endl;
			throw 0;
		}

		status = hiprandSetPseudoRandomGeneratorSeed(gen, std::chrono::high_resolution_clock::now().time_since_epoch().count());
		if (status != HIPRAND_STATUS_SUCCESS)
		{
			std::cerr << "hiprandSetPseudoRandomGeneratorSeed failed" << std::endl;
			throw 0;
		}

		hipblasStatus_t hipblasStatus_t = hipblasCreate(&handle);
		if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
		{
			std::cerr << "hipblasCreate failed" << std::endl;
			throw 0;
		}

		cuda_init = true;
	}
}

void cudaSafeMalloc(void** A, int n)
{
	hipError_t cudaStatus = hipMalloc(A, n);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void cudaSafeFree(void** A)
{
	if (A)
	{
		hipFree(*A);
		*A = NULL;
	}
}

void cudaCopyToDevice(void* _A, void* A, int n)
{
	hipError_t cudaStatus = hipMemcpy(_A, A, n, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMemcpy failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

__global__ void _matrixInitialize(float* A, int n, float r)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		A[i] = 2.0f * (A[i] - 0.5f) * r;
}

void matrixInitialize(float** A, int n, int m)
{
	hipError_t cudaStatus = hipMalloc(A, sizeof(float) * n * m);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void matrixInitialize(float** A, int n, int m, float r)
{
	matrixInitialize(A, n, m);

	hiprandStatus_t status = hiprandGenerateUniform(gen, *A, (size_t)n * (size_t)m);
	if (status != HIPRAND_STATUS_SUCCESS)
	{
		std::cerr << "hiprandGenerateUniform failed" << std::endl;
		throw 0;
	}

	int numBlocks = (n * m + blockSize - 1) / blockSize;
	_matrixInitialize<<<numBlocks, blockSize>>>(*A, n * m, r);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_matrixInitialize failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void matrixInitialize(float* A, int n, int m, float r)
{
	hiprandStatus_t status = hiprandGenerateUniform(gen, A, (size_t)n * (size_t)m);
	if (status != HIPRAND_STATUS_SUCCESS)
	{
		std::cerr << "hiprandGenerateUniform failed" << std::endl;
		throw 0;
	}

	int numBlocks = (n * m + blockSize - 1) / blockSize;
	_matrixInitialize<<<numBlocks, blockSize>>>(A, n * m, r);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_matrixInitialize failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void matrixFree(float** A)
{
	if (A)
	{
		hipFree(*A);
		*A = NULL;
	}
}

void matrixMultiply(float* C, float* A, int n0, int m0, float* B, int n1, int m1)
{
	if (m0 != n1)
		throw 0;

	float a = 1.0f;
	float b = 0.0f;

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n0, m1, m0, &a, A, n0, B, n1, &b, C, n0);
}

void matrixMultiplyTranspose(float* C, float* A, int n0, int m0, float* B, int n1, int m1)
{
	if (m0 != n1)
		throw 0;

	float a = 1.0f;
	float b = 0.0f;

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n0, m1, m0, &a, A, n0, B, m1, &b, C, n0);
}

void matrixTransposeMultiply(float* C, float* A, int n0, int m0, float* B, int n1, int m1)
{
	if (m0 != n1)
		throw 0;

	float a = 1.0f;
	float b = 0.0f;

	hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n0, m1, m0, &a, A, m0, B, n1, &b, C, n0);
}

__global__ void _arrayInitialize(float* A, int n, float r)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		A[i] = r;
}

void arrayInitialize(float** A, int n, float r)
{
	hipError_t cudaStatus = hipMalloc(A, sizeof(float) * n);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}

	int numBlocks = (n + blockSize - 1) / blockSize;
	_arrayInitialize<<<numBlocks, blockSize>>>(*A, n, r);
}

__global__ void _arrayAddRep(float* C, float* A, float* B, int n, int width)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		C[i] = A[i] + B[i % width];
}

__global__ void _arrayAddRepSigmoid(float* C, float* A, float* B, int n, int width)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		C[i] = 1.0f / (1.0f + expf(-(A[i] + B[i % width])));
}

__global__ void _arrayMultiplyDerivSigmoid(float* C, float* A, float* B, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		C[i] = A[i] * B[i] * (1.0f - B[i]);
}

void arrayCopyToDevice(float* _A, float* A, int n)
{
	hipError_t cudaStatus = hipMemcpy(_A, A, sizeof(float) * n, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMemcpy failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void arrayCopyToHost(float* A, float* _A, int n)
{
	//hipError_t cudaStatus = hipMemcpy(A, _A, sizeof(float) * n, hipMemcpyDeviceToHost);
	hipError_t cudaStatus = hipMemcpyAsync(A, _A, sizeof(float) * n, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMemcpy failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void arrayAddRep(float* C, float* A, float* B, int n, int m)
{
	int numBlocks = (n * m + blockSize - 1) / blockSize;
	_arrayAddRep<<<numBlocks, blockSize>>>(C, A, B, n * m, n);
}

void arrayAddRepSigmoid(float* C, float* A, float* B, int n, int m)
{
	int numBlocks = (n * m + blockSize - 1) / blockSize;
	_arrayAddRepSigmoid<<<numBlocks, blockSize>>>(C, A, B, n * m, n);
}

void arrayMultiplyDerivSigmoid(float* C, float* A, float* B, int n)
{
	int numBlocks = (n + blockSize - 1) / blockSize;
	_arrayMultiplyDerivSigmoid<<<numBlocks, blockSize>>>(C, A, B, n);
}

void arrayAbsSum(float* A, int n, float* out)
{
	hipblasSasum(handle, n, A, 1, out);
}

void arrayStep(float* B, float* A, float nu, int n)
{
	hipblasSaxpy(handle, n, &nu, A, 1, B, 1);
}

__global__ void _adamStep(float* A, float* B, float BETA1, float BETA2, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float dldw = A[i];
	float dldwm = (1.0f - BETA1) * dldw + BETA1 * B[i * 2];
	float dldwv = (1.0f - BETA2) * dldw * dldw + BETA2 * B[i * 2 + 1];
	B[i * 2] = dldwm;
	B[i * 2 + 1] = dldwv;
	A[i] = dldwm / (sqrtf(dldwv) + 1e-8);
}

void adamStep(float* A, float* B, float BETA1, float BETA2, int n)
{
	int numBlocks = (n + blockSize - 1) / blockSize;
	_adamStep<<<numBlocks, blockSize>>>(A, B, BETA1, BETA2, n);
}

__global__ void _potato(float* A, float* B, unsigned char* C, float* D, int* E)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ((i + 1) % 2500)
	{
		// _d3, _X3, _action + NUM_ACTIONS * offset, _reward + 2 * offset
		unsigned int m = i * NUM_ACTIONS;
		unsigned int n = m + NUM_ACTIONS;
		float q = 0.0f;
		int k = 0;
		unsigned char a;
		while ((a = C[m + k++]) < NUM_ACTIONS)
			if (B[n + a] > q)
				q = B[n + a];
		float x = B[m + E[i]];
		A[m + E[i]] = (x - (D[i * 2] + D[i * 2 + 1] * q)) * x * (1.0f - x);
	}
}

__global__ void _unpotato(float* A, int* B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ((i + 1) % 2500)
		A[i * NUM_ACTIONS + B[i]] = 0.0f;
}

void potato(float* A, float* B, unsigned char* C, float* D, int* E, int n)
{
	if (n % 1000)
		throw 0;
	_potato<<<n / 1000, 1000>>>(A, B, C, D, E);
}

void unpotato(float* A, int* B, int n)
{
	if (n % 1000)
		throw 0;
	_unpotato<<<n / 1000, 1000>>>(A, B);
}