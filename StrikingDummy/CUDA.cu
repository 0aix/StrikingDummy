#include "hip/hip_runtime.h"
#include "CUDA.cuh"
#include <iostream>
#include <chrono>
#include <math.h>
#include <hiprand.h>

int blockSize = 0;
bool cuda_init = false;

void cudaSafeDeviceSynchronize()
{
	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipDeviceSynchronize failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void cudaInitialize()
{
	if (!cuda_init)
	{
		hipError_t cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess)
		{
			std::cerr << "hipSetDevice failed: " << hipGetErrorString(cudaStatus) << std::endl;
			throw 0;
		}

		hipDeviceProp_t prop;
		cudaStatus = hipGetDeviceProperties(&prop, 0);
		if (cudaStatus != hipSuccess)
		{
			std::cerr << "hipGetDeviceProperties failed: " << hipGetErrorString(cudaStatus) << std::endl;
			throw 0;
		}
		blockSize = prop.maxThreadsPerBlock;
		if (blockSize != 1024)
			throw 0;

		//hipMemset(actions, 0, sizeof(actions));

		cuda_init = true;
	}
}

void cudaSafeMalloc(void** A, int n)
{
	hipError_t cudaStatus = hipMalloc(A, n);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void cudaSafeFree(void** A)
{
	if (A)
	{
		hipFree(*A);
		*A = NULL;
	}
}

void cudaCopyToDevice(void* _A, void* A, int n)
{
	hipError_t cudaStatus = hipMemcpy(_A, A, n, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMemcpy failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

__global__ void _matrixInitialize(float* A, int n, float r)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		A[i] = 2.0f * (A[i] - 0.5f) * r;
}

__global__ void _matrixMultiply(float* C, float* A, int n0, int m0, float* B, int n1, int m1)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float _A[32][32];
	__shared__ float _B[32][32];

	float sum = 0.0f;
	bool compute = i < n0 && j < m1;
	int Z = (m0 + 31) / 32;
	int K = (m0 + 31) % 32 + 1;

	for (int z = 0; z < Z; ++z)
	{
		int Ay = i;
		int Ax = threadIdx.x + z * 32;
		int By = threadIdx.y + z * 32;
		int Bx = j;
		if (Ay < n0 && Ax < m0)
			_A[threadIdx.y][threadIdx.x] = A[Ay + Ax * n0];
		if (By < n1 && Bx < m1)
			_B[threadIdx.y][threadIdx.x] = B[By + Bx * n1];

		__syncthreads();

		if (compute)
		{
			if (z == Z - 1)
			{
				for (int k = 0; k < K; ++k)
					sum += _A[threadIdx.y][k] * _B[k][threadIdx.x];
			}
			else
			{
#pragma unroll
				for (int k = 0; k < 32; ++k)
					sum += _A[threadIdx.y][k] * _B[k][threadIdx.x];
			}
		}

		__syncthreads();
	}
	if (compute)
		C[j * n0 + i] = sum;
}

__global__ void _matrixMultiplyTranspose(float* C, float* A, int n0, int m0, float* B, int n1, int m1)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float _A[32][32];
	__shared__ float _B[32][32];

	float sum = 0.0f;
	bool compute = i < n0 && j < m1;
	int Z = (m0 + 31) / 32;
	int K = (m0 + 31) % 32 + 1;

	for (int z = 0; z < Z; ++z)
	{
		int Ay = i;
		int Ax = threadIdx.x + z * 32;
		int By = threadIdx.y + z * 32;
		int Bx = j;
		if (Ay < n0 && Ax < m0)
			_A[threadIdx.y][threadIdx.x] = A[Ay + Ax * n0];
		if (By < n1 && Bx < m1)
			_B[threadIdx.y][threadIdx.x] = B[By * m1 + Bx];

		__syncthreads();

		if (compute)
		{
			if (z == Z - 1)
			{
				for (int k = 0; k < K; ++k)
					sum += _A[threadIdx.y][k] * _B[k][threadIdx.x];
			}
			else
			{
#pragma unroll
				for (int k = 0; k < 32; ++k)
					sum += _A[threadIdx.y][k] * _B[k][threadIdx.x];
			}
		}

		__syncthreads();
	}
	if (compute)
		C[j * n0 + i] = sum;
}

__global__ void _matrixTranspose(float* B, float* A, int n, int m)
{
	__shared__ float tile[32][33];

	int i = blockIdx.y * 32 + threadIdx.y;
	int j = blockIdx.x * 32 + threadIdx.x;
	if (i < n && j < m)
		tile[threadIdx.y][threadIdx.x] = A[i * m + j];

	__syncthreads();

	i = blockIdx.x * 32 + threadIdx.y;
	j = blockIdx.y * 32 + threadIdx.x;
	if (i < m && j < n)
		B[i * n + j] = tile[threadIdx.x][threadIdx.y];
	/*
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n && j < m)
		B[i * m + j] = A[j * n + i];
	*/
}

void matrixInitialize(float** A, int n, int m)
{
	hipError_t cudaStatus = hipMalloc(A, sizeof(float) * n * m);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void matrixInitialize(float** A, int n, int m, float r)
{
	matrixInitialize(A, n, m);

	hiprandGenerator_t gen;
	
	hiprandStatus_t status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);
	if (status != HIPRAND_STATUS_SUCCESS)
	{
		std::cerr << "hiprandCreateGenerator failed" << std::endl;
		throw 0;
	}

	status = hiprandSetPseudoRandomGeneratorSeed(gen, std::chrono::high_resolution_clock::now().time_since_epoch().count());
	if (status != HIPRAND_STATUS_SUCCESS)
	{
		std::cerr << "hiprandSetPseudoRandomGeneratorSeed failed" << std::endl;
		throw 0;
	}

	status = hiprandGenerateUniform(gen, *A, (size_t)n * (size_t)m);
	if (status != HIPRAND_STATUS_SUCCESS)
	{
		std::cerr << "hiprandGenerateUniform failed" << std::endl;
		throw 0;
	}

	status = hiprandDestroyGenerator(gen);
	if (status != HIPRAND_STATUS_SUCCESS)
	{
		std::cerr << "hiprandDestroyGenerator failed" << std::endl;
		throw 0;
	}

	int numBlocks = (n * m + blockSize - 1) / blockSize;
	_matrixInitialize<<<numBlocks, blockSize>>>(*A, n * m, r);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_matrixInitialize failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void matrixFree(float** A)
{
	if (A)
	{
		hipFree(*A);
		*A = NULL;
	}
}

void matrixMultiply(float* C, float* A, int n0, int m0, float* B, int n1, int m1)
{
	if (m0 != n1)
		throw 0;

	dim3 numThreads(32, 32);
	dim3 numBlocks((m1 + 31) / 32, (n0 + 31) / 32);

	_matrixMultiply<<<numBlocks, numThreads>>>(C, A, n0, m0, B, n1, m1);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_matrixMultiply failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void matrixMultiplyTranspose(float* C, float* A, int n0, int m0, float* B, int n1, int m1)
{
	if (m0 != n1)
		throw 0;

	dim3 numThreads(32, 32);
	dim3 numBlocks((m1 + 31) / 32, (n0 + 31) / 32);

	_matrixMultiplyTranspose<<<numBlocks, numThreads>>>(C, A, n0, m0, B, n1, m1);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_matrixMultiply failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void matrixTranspose(float* B, float* A, int n, int m)
{
	dim3 numThreads(32, 32);
	dim3 numBlocks((m + 31) / 32, (n + 31) / 32);

	_matrixTranspose<<<numBlocks, numThreads>>>(B, A, n, m);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_matrixTranspose failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

__global__ void _arrayAdd(float* C, float* A, float* B, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		C[i] = A[i] + B[i];
}

__global__ void _arrayAddScalar(float* C, float* A, float b, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		C[i] = A[i] + b;
}

__global__ void _arrayAddRep(float* C, float* A, float* B, int n, int width)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		C[i] = A[i] + B[i % width];
}

__global__ void _arraySubtract(float* C, float* A, float* B, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		C[i] = A[i] - B[i];
}

__global__ void _arrayMultiply(float* C, float* A, float* B, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		C[i] = A[i] * B[i];
}

__global__ void _arrayMultiplyScalar(float* B, float* A, float b, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		B[i] = A[i] * b;
}

__global__ void _arrayDivide(float* C, float* A, float* B, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		C[i] = A[i] / B[i];
}

__global__ void _arraySigmoid(float* B, float* A, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		B[i] = 1.0f / (1.0f + expf(-A[i]));
}

__global__ void _arrayDerivSigmoid(float* B, float* A, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		B[i] = A[i] * (1.0f - A[i]);
}

__global__ void _arrayReLU(float* B, float* A, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		B[i] = fmaxf(0.0f, A[i]);
}

__global__ void _arrayDerivReLU(float* B, float* A, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		B[i] = A[i] > 0.0f ? 1.0f : 0.0f;
}

__global__ void _arraySqrt(float* B, float* A, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		B[i] = sqrtf(A[i]);
}

void arrayCopyToDevice(float* _A, float* A, int n)
{
	hipError_t cudaStatus = hipMemcpy(_A, A, sizeof(float) * n, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMemcpy failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void arrayCopyToHost(float* A, float* _A, int n)
{
	hipError_t cudaStatus = hipMemcpy(A, _A, sizeof(float) * n, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "hipMemcpy failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void arrayAdd(float* C, float* A, float* B, int n)
{
	int numBlocks = (n + blockSize - 1) / blockSize;
	_arrayAdd<<<numBlocks, blockSize>>>(C, A, B, n);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_arrayAdd failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void arrayAdd(float* C, float* A, float b, int n)
{
	int numBlocks = (n + blockSize - 1) / blockSize;
	_arrayAddScalar<<<numBlocks, blockSize>>>(C, A, b, n);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_arrayAddScalar failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void arrayAddRep(float* C, float* A, float* B, int n, int m)
{
	int numBlocks = (n * m + blockSize - 1) / blockSize;
	_arrayAddRep<<<numBlocks, blockSize>>>(C, A, B, n * m, n);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_arrayAddRep failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void arraySubtract(float* C, float* A, float* B, int n)
{
	int numBlocks = (n + blockSize - 1) / blockSize;
	_arraySubtract<<<numBlocks, blockSize>>>(C, A, B, n);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_arraySubtract failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void arrayMultiply(float* C, float* A, float* B, int n)
{
	int numBlocks = (n + blockSize - 1) / blockSize;
	_arrayMultiply<<<numBlocks, blockSize>>>(C, A, B, n);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_arrayMultiply failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void arrayMultiply(float* C, float* A, float b, int n)
{
	int numBlocks = (n + blockSize - 1) / blockSize;
	_arrayMultiplyScalar<<<numBlocks, blockSize>>>(C, A, b, n);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_arrayMultiplyScalar failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void arrayDivide(float* C, float* A, float* B, int n)
{
	int numBlocks = (n + blockSize - 1) / blockSize;
	_arrayDivide<<<numBlocks, blockSize>>>(C, A, B, n);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_arrayDivide failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void arraySigmoid(float* B, float* A, int n)
{
	int numBlocks = (n + blockSize - 1) / blockSize;
	_arraySigmoid<<<numBlocks, blockSize>>>(B, A, n);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_arraySigmoid failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void arrayDerivSigmoid(float* B, float* A, int n)
{
	int numBlocks = (n + blockSize - 1) / blockSize;
	_arrayDerivSigmoid<<<numBlocks, blockSize>>>(B, A, n);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_arrayDerivSigmoid failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void arrayReLU(float* B, float* A, int n)
{
	int numBlocks = (n + blockSize - 1) / blockSize;
	_arrayReLU<<<numBlocks, blockSize>>>(B, A, n);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_arrayReLU failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void arrayDerivReLU(float* B, float* A, int n)
{
	int numBlocks = (n + blockSize - 1) / blockSize;
	_arrayDerivReLU<<<numBlocks, blockSize>>>(B, A, n);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_arrayDerivReLU failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

void arraySqrt(float* B, float* A, int n)
{
	int numBlocks = (n + blockSize - 1) / blockSize;
	_arraySqrt<<<numBlocks, blockSize>>>(B, A, n);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_arraySqrt failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
	//cudaSafeDeviceSynchronize();
}

__global__ void _unpotato(float* A, int* B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ((i + 1) % 2500)
		A[i * 20 + B[i]] = 0.0f;
}

__global__ void _potato(float* A, float* B, bool* C, float* D, int* E)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ((i + 1) % 2500)
	{
		// _d3, _X3, _action + 20 * offset, _reward + 2 * offset
		int j;
		float q = 0.0f;
#pragma unroll
		for (int k = 0; k < 20; ++k)
		{
			if (C[i * 20 + k] && B[(i + 1) * 20 + k] > q)
			{
				j = k;
				q = B[(i + 1) * 20 + k];
			}
		}
		float x = B[i * 20 + E[i]];
		A[i * 20 + E[i]] = (x - (D[i * 2] + D[i * 2 + 1] * q)) * x * (1.0f - x);
	}
}

void unpotato(float* A, int* B)
{
	_unpotato<<<10000 / 1000, 1000>>>(A, B);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_unpotato failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}

void potato(float* A, float* B, bool* C, float* D, int* E)
{
	_potato<<<10000 / 1000, 1000>>>(A, B, C, D, E);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cerr << "_potato failed: " << hipGetErrorString(cudaStatus) << std::endl;
		throw 0;
	}
}